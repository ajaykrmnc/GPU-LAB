#include "hip/hip_runtime.h"
#include "Helper.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ int index_calculate(int row, int col, int cols) {
  return col + row * cols;
}

__global__ void matTransposePart(float *mat, float *res, int rows, int cols) {
	int index = index_calculate(blockIdx.x, blockIdx.y, gridDim.x);
	int transpose_index = index_calculate(blockIdx.y, blockIdx.x, gridDim.x);
	if (index < (rows * cols) && transpose_index < (rows * cols)) {
		res[transpose_index] = mat[index];
	}
}

__global__ void matMulPart(float *mat1, float *mat2, float *res, int rows,
                           int cols) {
  int index = index_calculate(blockIdx.x, blockIdx.y, gridDim.x);
  if (index < (rows * cols)) {
    float temp = 0;
    for (int i = 0; i < cols; i++) {
      temp += mat1[index_calculate(i, blockIdx.y, rows)] *
              mat2[index_calculate(blockIdx.x, i, cols)];
    }
    res[index] = temp;
  }
}

void matrixTranspose(float *mat, float *res, int rows, int cols) {
  int mat_bytes = rows * cols * sizeof(float);
  dim3 grid(cols, rows, 1);
  dim3 block(1);

  float *dev_mat, *dev_res;

  hipMalloc((void **)&dev_mat, mat_bytes);
  hipMalloc((void **)&dev_res, mat_bytes);

  hipMemcpy(dev_mat, mat, mat_bytes, hipMemcpyHostToDevice);

  matTransposePart<<<grid, block>>>(dev_mat, dev_res, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, dev_res, mat_bytes, hipMemcpyDeviceToHost);

  hipFree(dev_mat);
  hipFree(dev_res);

  hipDeviceReset();
}

void matrixMul(float *mat1, float *mat2, float *res, int rows, int cols) {
  int mat_bytes = rows * cols * sizeof(float);
  dim3 grid(cols, rows, 1);
  dim3 block(1);

  float *dev_mat1, *dev_mat2, *dev_res;

  hipMalloc((void **)&dev_mat1, mat_bytes);
  hipMalloc((void **)&dev_mat2, mat_bytes);
  hipMalloc((void **)&dev_res, mat_bytes);

  hipMemcpy(dev_mat1, mat1, mat_bytes, hipMemcpyHostToDevice);
  hipMemcpy(dev_mat2, mat2, mat_bytes, hipMemcpyHostToDevice);

  matMulPart<<<grid, block>>>(dev_mat1, dev_mat2, dev_res, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, dev_res, mat_bytes, hipMemcpyDeviceToHost);

  hipFree(dev_mat1);
  hipFree(dev_mat2);
  hipFree(dev_res);

  hipDeviceReset();
}

