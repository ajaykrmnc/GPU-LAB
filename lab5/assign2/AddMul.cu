#include "hip/hip_runtime.h"
#include "AddMul.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ int index_calculate(int row, int col, int cols) {
  return col + row * cols;
}

__global__ void matAddPart(float *mat1, float *mat2, float *res, int rows,
                           int cols) {
  int index = index_calculate(blockIdx.x, blockIdx.y, gridDim.x);
  if (index < (rows * cols)) {
    res[index] = mat1[index] + mat2[index];
  }
}

__global__ void matMulPart(float *mat1, float *mat2, float *res, int rows,
                           int cols) {
  int index = index_calculate(blockIdx.x, blockIdx.y, gridDim.x);
  if (index < (rows * cols)) {
    float temp = 0;
    for (int i = 0; i < cols; i++) {
      temp += mat1[index_calculate(i, blockIdx.y, rows)] *
              mat2[index_calculate(blockIdx.x, i, cols)];
    }
    res[index] = temp;
  }
}

void matrixAdd(float *mat1, float *mat2, float *res, int rows, int cols) {
  int mat_bytes = rows * cols * sizeof(float);
  dim3 grid(cols, rows, 1);
  dim3 block(1);

  float *dev_mat1, *dev_mat2, *dev_res;

  hipMalloc((void **)&dev_mat1, mat_bytes);
  hipMalloc((void **)&dev_mat2, mat_bytes);
  hipMalloc((void **)&dev_res, mat_bytes);

  hipMemcpy(dev_mat1, mat1, mat_bytes, hipMemcpyHostToDevice);
  hipMemcpy(dev_mat2, mat2, mat_bytes, hipMemcpyHostToDevice);

  matAddPart<<<grid, block>>>(dev_mat1, dev_mat2, dev_res, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, dev_res, mat_bytes, hipMemcpyDeviceToHost);

  hipFree(dev_mat1);
  hipFree(dev_mat2);
  hipFree(dev_res);

  hipDeviceReset();
}

void matrixMul(float *mat1, float *mat2, float *res, int rows, int cols) {
  int mat_bytes = rows * cols * sizeof(float);
  dim3 grid(cols, rows, 1);
  dim3 block(1);

  float *dev_mat1, *dev_mat2, *dev_res;

  hipMalloc((void **)&dev_mat1, mat_bytes);
  hipMalloc((void **)&dev_mat2, mat_bytes);
  hipMalloc((void **)&dev_res, mat_bytes);

  hipMemcpy(dev_mat1, mat1, mat_bytes, hipMemcpyHostToDevice);
  hipMemcpy(dev_mat2, mat2, mat_bytes, hipMemcpyHostToDevice);

  matMulPart<<<grid, block>>>(dev_mat1, dev_mat2, dev_res, rows, cols);
  hipDeviceSynchronize();

  hipMemcpy(res, dev_res, mat_bytes, hipMemcpyDeviceToHost);

  hipFree(dev_mat1);
  hipFree(dev_mat2);
  hipFree(dev_res);

  hipDeviceReset();
}

